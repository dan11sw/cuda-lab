#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// Ядро CUDA (выполняется на GPU)
__global__ void helloFromGPU() {
    printf("Hello World from GPU! (Thread %d)\n", threadIdx.x);
}

int main() {
    printf("Hello World from CPU!\n");

    // Запуск ядра на GPU (1 блок, 10 потоков)
    helloFromGPU << <1, 10 >> > ();

    // Синхронизация и проверка ошибок
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(hipGetLastError()));
        return 1;
    }

    system("PAUSE");

    return 0;
}