#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "./common/book.h"
#include <stdio.h>

__global__ void add(int a, int b, int* c) {
	*c = a + b;
}

int main(void) {
	int c;
	int* dev_c;

	// Выделяем память на устройстве GPU с помощью метода hipMalloc
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

	// Вызываем функцию add на устройстве GPU, передавая ей в качестве аргумента выделенную память под GPU устройство
	add<<<1, 1>>>(2, 7, dev_c);

	// Осуществляем перенос данных из указателя размещённом в памяти GPU в указатель размещённый в CPU
	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost));

	printf("2 + 7 = %d\n", c);

	// Освобождение занимаемой памяти в GPU
	hipFree(dev_c);
	system("pause");
	return 0;
}