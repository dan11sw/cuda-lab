#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "./common/book.h"
#include <stdio.h>
#include <vector>
#include <iostream>

// Ядро CUDA для вывода элементов массива
__global__ void print_vector(int* arr, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
    if (idx < size) {
        printf("GPU: arr[%d] = %d\n", idx, arr[idx]); // Вывод из GPU
    }
}

int main() {
    const int N = 10;
	// CPU array
    std::vector<int> host_array(N, 0);

	// GPU array
    int *dev_array;

    // Инициализируем массив CPU
    for (int i = 0; i < N; i++) {
        host_array[i] += i;
    }

	// Выделение памяти на GPU
    HANDLE_ERROR(hipMalloc((void**)&dev_array, N * sizeof(int)));

    // Копируем данные из CPU массива в GPU массив
    HANDLE_ERROR(hipMemcpy(dev_array, host_array.data(), N * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice));

    // Запускаем ядро для вывода массива на GPU
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    print_vector<<<gridSize, blockSize>>>(dev_array, N);

    // Ожидаем завершения работы ядра
    hipDeviceSynchronize();

    // Освобождаем занятую память
    hipFree(dev_array);

	system("PAUSE");
    return 0;
}