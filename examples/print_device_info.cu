#include "hip/hip_runtime.h"
#include ""
#include "./common/book.h"
#include <stdio.h>
#include <vector>
#include <iostream>

int main() {
	hipDeviceProp_t prop;

	int count = 0;

	// Получаем кол-во устройств, которые поддерживают архитектуру CUDA
	HANDLE_ERROR(hipGetDeviceCount(&count));

	for(int i = 0; i < count; i++) {
		// Получение информации об устройстве
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));

		// Вывод информации об устройстве
		std::cout << "Name: " << prop.name << std::endl;
		std::cout << "TotalGlobalMem (Gb): " << prop.totalGlobalMem / (1024.0 * 1024 * 1024) << std::endl;
		std::cout << "SharedMemPerBlock: " << prop.sharedMemPerBlock << std::endl; 
		std::cout << "WarpSize: " << prop.warpSize << std::endl;
		std::cout << "MaxThreadsPerBlock: " << prop.maxThreadsPerBlock << std::endl << std::endl;
	}

	system("PAUSE");
    return 0;
}